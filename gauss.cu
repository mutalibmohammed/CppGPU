
#include <hip/hip_runtime.h>
#include <iostream>



template <typename T>
__global__
void gauss_seidel(int ny, int nx, const T *p, T *pnew) {
    for (int i = 1; i < ny - 1; i++) {
        for (int j = 1; j < nx - 1; j++) {
            printf("Do I xome here\n");
            pnew[i * nx + j] = 0.25 * (pnew[(i - 1) * nx + j] + pnew[i * nx + (j - 1)] + p[(i + 1) * nx + j] + p[i * nx + (j + 1)]);
        }
    }
}

int main() {

    typedef double gtype;

    constexpr const int iterations = 1000;
    constexpr const int nx = 101;
    constexpr const int ny = 101;
 
    auto p = new gtype[ny][nx]();
    auto pnew = new gtype[ny][nx]();
 
    gtype *d_p;
    gtype *d_pnew;

    hipError_t err;

    err = hipMalloc((void **) &d_p, ny * nx * sizeof(gtype));
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;

    err = hipMalloc((void **) &d_pnew, ny * nx * sizeof(gtype));
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;


    for (int i = 1; i < ny - 1; i++)
    {
        for (int j = 1; j < nx - 1; j++)
        {
            p[i][j] = 1;
            pnew[i][j] = 1;
        }
    }

    
    err = hipMemcpy(d_p, p, ny * nx * sizeof(gtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(d_pnew, pnew, ny * nx * sizeof(gtype), hipMemcpyHostToDevice);
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;


    for (int i = 0; i < iterations; i++)
    {
        gauss_seidel<<<1, 1>>>(ny, nx, (i % 2 == 0) ? d_p: d_pnew, (i % 2 == 0) ? d_pnew : d_p);
        hipError_t errSync  = hipGetLastError();
        hipError_t errAsync = hipDeviceSynchronize();
        if (errSync != hipSuccess) 
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
        if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    }

    err = hipMemcpy(p, d_p, ny * nx * sizeof(gtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(pnew, d_pnew, ny * nx * sizeof(gtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess) std::cout << hipGetErrorString(err) << std::endl;


    delete[] p;
    delete[] pnew;

    hipFree(d_p);
    hipFree(d_pnew);
    return 0;
}