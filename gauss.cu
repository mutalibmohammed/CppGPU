#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>

template <typename T>
__global__ void gauss_seidel(int ny, int nx, const T *p, T *pnew)
{
    for (int i = 1; i < ny - 1; i++)
    {
        for (int j = 1; j < nx - 1; j++)
        {
            pnew[i * nx + j] = 0.25 * (pnew[(i - 1) * nx + j] + pnew[i * nx + (j - 1)] + p[(i + 1) * nx + j] + p[i * nx + (j + 1)]);
        }
    }
}

template <typename T>
__global__ void gauss_seidel_wave(int ny, int nx, const T *p, T *pnew)
{

    for (int wavefront = 2; wavefront < ny + nx - 1; wavefront++)
    {

        int xmin = max(1, wavefront - ((ny - 1) - 1));
        // TODO: maybe change it so its less than or equal to.
        int xmax = min(wavefront, nx - 1);

        int x = threadIdx.x;

        if (x >= xmin && x < xmax)
        {
            int y = wavefront - x;
            pnew[y * nx + x] = 0.25 * (pnew[(y - 1) * nx + x] + pnew[y * nx + (x - 1)] + p[(y + 1) * nx + x] + p[y * nx + (x + 1)]);
        }

        __syncthreads();
    }
}

template <typename T>
__global__ void gauss_seidel_block_wave(int nby, int nbx, int ny, int nx, const T *p, T *pnew, int bwavefront)
{
    // Given blockid.x and bwavefront, calculate the startx and starty
    // BlockIdx.x represents the Number of the block on thw wavefront. O is the leftmost block on the wavefront.
    int bxmin = max(0, bwavefront - ((ny / nby - 1)));
    int bxmax = min(bwavefront, nx / nbx - 1);

    int bx = blockIdx.x + bxmin;
    int by = bwavefront - bx;

    // printf("B Wavefront: %d, Block(%2d): %d, %d\n", bwavefront, blockIdx bx, by);

    if (bx > bxmax)
    {
        printf("Block out of bounds\n");
    }

    int startx = bx * nbx;
    int starty = by * nby;

    // TODO: Handle block boundaries

    for (int wavefront = 0; wavefront < nby + nbx - 1; wavefront++)
    {

        int xmin = max(1, wavefront - ((nby - 1) - 1));
        int xmax = min(wavefront, nbx - 1);

        int x = threadIdx.x;

        if (x >= xmin && x <= xmax)
        {
            int y = wavefront - x;
            y = y + starty;
            x = x + startx;
            if(x != 0 && x != nx-1 && y != 0 && y != ny-1)
                pnew[y * nx + x] = 0.25 * (pnew[(y - 1) * nx + x] + pnew[y * nx + (x - 1)] + p[(y + 1) * nx + x] + p[y * nx + (x + 1)]);
        }

        __syncthreads();
    }
}

int main()
{

    typedef double gtype;

    constexpr const int iterations = 10000;
    constexpr const int nx = 100;
    constexpr const int ny = 100;

    auto p = new gtype[ny][nx]();
    auto pnew = new gtype[ny][nx]();

    gtype *d_p;
    gtype *d_pnew;

    hipError_t err;

    err = hipMalloc((void **)&d_p, ny * nx * sizeof(gtype));
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    err = hipMalloc((void **)&d_pnew, ny * nx * sizeof(gtype));
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    // Dirichlet boundary conditions
    for (int y = 0; y < ny; y++)
    {
        p[y][0] = 10;
        pnew[y][0] = 10;
        p[y][nx - 1] = 10;
        pnew[y][nx - 1] = 10;
    }

    for (int x = 0; x < nx; x++)
    {
        p[0][x] = 10;
        pnew[0][x] = 10;
        p[ny - 1][x] = 10;
        pnew[ny - 1][x] = 10;
    }

    err = hipMemcpy(d_p, p, ny * nx * sizeof(gtype), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(d_pnew, pnew, ny * nx * sizeof(gtype), hipMemcpyHostToDevice);
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    int nbx = 20;
    int nby = 20;

    for (int i = 0; i < iterations; i++)
    {
        // gauss_seidel<<<1, std::max(nx, ny)>>>(ny, nx, (i % 2 == 0) ? d_p : d_pnew, (i % 2 == 0) ? d_pnew : d_p);
        // hipError_t errSync = hipGetLastError();
        // hipError_t errAsync = hipDeviceSynchronize();
        // if (errSync != hipSuccess)
        //     printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
        // if (errAsync != hipSuccess)
        //     printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

        for(int bwavefront = 0; bwavefront < ny/nby + nx/nbx - 1; bwavefront++) {

            // Figure out the number of blocks on the wavefront
            int bxmin = max(0, bwavefront - ((ny/nby - 1) ));
            int bxmax = min(bwavefront, nx/nbx - 1);
            int num_blocks = bxmax - bxmin + 1;

            // Call the kernel with the number of blocks
            gauss_seidel_block_wave<<<num_blocks, std::min(nbx, nby)>>>(nby, nbx, ny, nx, (i % 2 == 0) ? d_p : d_pnew, (i % 2 == 0) ? d_pnew : d_p, bwavefront);

            hipError_t errSync = hipGetLastError();
            hipError_t errAsync = hipDeviceSynchronize();
            if (errSync != hipSuccess)
                printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
            if (errAsync != hipSuccess)
                printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

            }
    }

    err = hipMemcpy(p, d_p, ny * nx * sizeof(gtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    err = hipMemcpy(pnew, d_pnew, ny * nx * sizeof(gtype), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        std::cout << hipGetErrorString(err) << std::endl;

    gtype sum = 0;
    for (int y = 0; y < ny; y++)
    {
        for (int x = 0; x < nx; x++)
        {
            sum += p[y][x];
        }
    }
    std::cout << "Sum: " << sum << std::endl;
    delete[] p;
    delete[] pnew;

    hipFree(d_p);
    hipFree(d_pnew);
    return 0;
}